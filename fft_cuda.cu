// nvcc -o fft_cuda fft_cuda.cu -arch=compute_75 -code=sm_75 -use_fast_math -O3
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <complex.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <hip/hip_complex.h>

__device__ int reverseBits(int num, int log2n) {
    int reversed = 0;
    for (int i = 0; i < log2n; i++) {
        reversed = (reversed << 1) | (num & 1);
        num >>= 1;
    }
    return reversed;
}

__global__ void bitReverseCopy(hipDoubleComplex* a, hipDoubleComplex* b, int n, int log2n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        int rev = reverseBits(tid, log2n);
        b[rev] = a[tid];
    }
}

__global__ void fftKernel(hipDoubleComplex* a, int n, bool invert) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;;

    for (int len = 2; len <= n; len <<= 1) {
        double ang = 2 * 3.1415926 * (invert ? -1 : 1) / (double)len;
        hipDoubleComplex wlen = make_hipDoubleComplex(cos(ang), sin(ang));
        for (int i = tid; i < n; i += numThreads * len) {
            hipDoubleComplex w = make_hipDoubleComplex(1, 0);
            for (int j = 0; j < len>>1; j++) {
                hipDoubleComplex u = a[i + j];
                hipDoubleComplex v = hipCmul(a[i + j + len>>1], w);
                a[i + j] = hipCadd(u, v);
                a[i + j + len>>1] = hipCsub(u, v);
                w = hipCmul(w, wlen);
            }
        }
    }
}

__global__ void normalize(hipDoubleComplex* a, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        a[index] = hipCdiv(a[index], make_hipDoubleComplex(n, 0));
    }
}

void fft(hipDoubleComplex *h_a, int n, bool invert) {
    hipDoubleComplex *d_a;
    hipMalloc(&d_a, n * sizeof(hipDoubleComplex));
    hipMemcpy(d_a, h_a, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    int log2n = log2((double)n);
    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);

    hipDoubleComplex *d_temp;
    hipMalloc(&d_temp, n * sizeof(hipDoubleComplex));
    bitReverseCopy<<<grid, block>>>(d_a, d_temp, n, log2n);
    hipMemcpy(d_a, d_temp, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

    fftKernel<<<grid, block>>>(d_a, n, invert);

    if (invert) {
        normalize<<<grid, block>>>(d_a, n);
    }

    hipMemcpy(h_a, d_a, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_temp);
}

// Function to generate complex data
void generateComplexData(hipDoubleComplex *a, int n) {
    for (int i = 0; i < n; ++i) {
        double real = static_cast<double>(rand()) / RAND_MAX;
        double imag = static_cast<double>(rand()) / RAND_MAX;
        a[i] = make_hipDoubleComplex(real, imag);
    }
}

int main() {
    int sizes[] = {256, 512, 1024,2048,4096,8192,16384,32768,65536,131072,262144,524288,1048576,2097152,4194304,8388608};
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);

    // Initialize cuda
    hipFree(0);

    for (int i = 0; i < num_sizes; ++i) {
        int n = sizes[i];
        hipDoubleComplex *data = (hipDoubleComplex *)malloc(n * sizeof(hipDoubleComplex));

        // Generate data
        generateComplexData(data, n);

        // Measure start time
        struct timeval start, end;
        gettimeofday(&start, NULL);
        
        // Run FFT
        fft(data, n, false);
        
        // Measure end time
        gettimeofday(&end, NULL);
        long seconds = (end.tv_sec - start.tv_sec);
        long micros = ((seconds * 1000000) + end.tv_usec) - (start.tv_usec);

        printf("FFT size %d - Execution time: %ld microseconds\n", n, micros);

        free(data);
    }

    return 0;
}