#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <complex.h>
#include <hip/hip_runtime.h>

#include <hip/hip_complex.h>


#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int reverseBits(int num, int log2n) {
    int reversed = 0;
    for (int i = 0; i < log2n; i++) {
        reversed = (reversed << 1) | (num & 1);
        num >>= 1;
    }
    return reversed;
}

__global__ void bitReverseCopy(hipDoubleComplex* a, hipDoubleComplex* b, int n, int log2n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        int rev = reverseBits(tid, log2n);
        b[rev] = a[tid];
    }
}

__global__ void fftKernel(hipDoubleComplex* a, int n, bool invert) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;

    extern __shared__ hipDoubleComplex sdata[];

    for (int len = 2; len <= n; len <<= 1) {
        double ang = 2 * M_PI * (invert ? -1 : 1) / (double)len;
        hipDoubleComplex wlen = make_hipDoubleComplex(cos(ang), sin(ang));
        for (int i = tid; i < n; i += numThreads * len) {
            hipDoubleComplex w = make_hipDoubleComplex(1, 0);
            for (int j = 0; j < len / 2; j++) {
                sdata[j] = a[i + j + len / 2];
                __syncthreads();  // Ensure all data is loaded

                hipDoubleComplex u = a[i + j];
                hipDoubleComplex v = hipCmul(sdata[j], w);
                a[i + j] = hipCadd(u, v);
                a[i + j + len / 2] = hipCsub(u, v);
                w = hipCmul(w, wlen);
                
                __syncthreads();  // Sync before next usage of shared memory
            }
        }
    }
}

__global__ void normalize(hipDoubleComplex* a, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        a[index] = hipCdiv(a[index], make_hipDoubleComplex(n, 0));
    }
}

void fft(hipDoubleComplex *h_a, int n, bool invert) {
    hipDoubleComplex *d_a;
    hipMalloc(&d_a, n * sizeof(hipDoubleComplex));
    hipMemcpy(d_a, h_a, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    int log2n = log2((double)n);
    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);

    hipDoubleComplex *d_temp;
    hipMalloc(&d_temp, n * sizeof(hipDoubleComplex));
    bitReverseCopy<<<grid, block>>>(d_a, d_temp, n, log2n);
    hipMemcpy(d_a, d_temp, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

    fftKernel<<<grid, block, n * sizeof(hipDoubleComplex) / 2>>>(d_a, n, invert);  // Allocate shared memory

    if (invert) {
        normalize<<<grid, block>>>(d_a, n);
    }
    cudaCheckError(hipMalloc(&d_a, n * sizeof(hipDoubleComplex)));
    hipMemcpy(h_a, d_a, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_temp);
}

void generateComplexData(hipDoubleComplex *a, int n) {
    for (int i = 0; i < n; ++i) {
        double real = static_cast<double>(rand()) / RAND_MAX;
        double imag = static_cast<double>(rand()) / RAND_MAX;
        a[i] = make_hipDoubleComplex(real, imag);
    }
}

int main() {
    int sizes[] = {256, 512, 1024, 2048, 4096, 8192, 16384, 32768, 65536, 131072, 262144, 524288, 1048576, 2097152, 4194304, 8388608};
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);

    hipFree(0);  // Initialize CUDA

    for (int i = 0; i < num_sizes; ++i) {
        int n = sizes[i];
        hipDoubleComplex *data = (hipDoubleComplex *)malloc(n * sizeof(hipDoubleComplex));

        generateComplexData(data, n);

        struct timeval start, end;
        gettimeofday(&start, NULL);
        
        fft(data, n, false);
        
        gettimeofday(&end, NULL);
        long seconds = (end.tv_sec - start.tv_sec);
        long micros = ((seconds * 1000000) + end.tv_usec) - (start.tv_usec);

        printf("FFT size %d - Execution time: %ld microseconds\n", n, micros);

        free(data);
    }

    return 0;
}
