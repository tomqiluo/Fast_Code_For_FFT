#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <complex.h>
#include <hip/hip_runtime.h>

#include <hip/hip_complex.h>

__device__ int reverseBits(int num, int log2n) {
    int reversed = 0;
    for (int i = 0; i < log2n; i++) {
        reversed = (reversed << 1) | (num & 1);
        num >>= 1;
    }
    return reversed;
}

__global__ void bitReverseCopy(hipDoubleComplex* a, hipDoubleComplex* b, int n, int log2n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        int rev = reverseBits(tid, log2n);
        b[rev] = a[tid];
    }
}

__global__ void fftKernel(hipDoubleComplex* a, int n, bool invert) {
    extern __shared__ hipDoubleComplex temp[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;

    for (int len = 2; len <= n; len <<= 1) {
        int halfLen = len / 2;
        double ang = 2 * M_PI * (invert ? -1 : 1) / len;
        hipDoubleComplex wlen = make_hipDoubleComplex(cos(ang), sin(ang));

        for (int i = tid; i < n; i += numThreads * len) {
            for (int j = 0; j < halfLen; ++j) {
                int index1 = i + j;
                int index2 = i + j + halfLen;

                // Load the data into shared memory
                temp[threadIdx.x * 2] = a[index1];
                temp[threadIdx.x * 2 + 1] = a[index2];

                __syncthreads(); // Make sure all writes to shared memory are done

                hipDoubleComplex u = temp[threadIdx.x * 2];
                hipDoubleComplex t = temp[threadIdx.x * 2 + 1];
                hipDoubleComplex w = make_hipDoubleComplex(1, 0);
                
                // Perform the twiddle factor multiplication on a single element
                hipDoubleComplex twiddled = hipCmul(w, t);

                // Save the results back into global memory
                a[index1] = hipCadd(u, twiddled);
                a[index2] = hipCsub(u, twiddled);

                // Update the twiddle factor
                if (j != 0) {
                    w = hipCmul(w, wlen);
                }
                
                __syncthreads(); // Sync before the next iteration
            }
        }
    }
}

__global__ void normalize(hipDoubleComplex* a, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        a[index] = hipCdiv(a[index], make_hipDoubleComplex(n, 0));
    }
}

void fft(hipDoubleComplex *h_a, int n, bool invert) {
    hipDoubleComplex *d_a;
    hipMalloc(&d_a, n * sizeof(hipDoubleComplex));
    hipMemcpy(d_a, h_a, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    int log2n = log2((double)n);
    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);

    hipDoubleComplex *d_temp;
    hipMalloc(&d_temp, n * sizeof(hipDoubleComplex));
    bitReverseCopy<<<grid, block>>>(d_a, d_temp, n, log2n);
    hipMemcpy(d_a, d_temp, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

    int sharedSize = sizeof(hipDoubleComplex) * block.x * 2; // Allocate twice the block size for pair storage
    fftKernel<<<grid, block, sharedSize>>>(d_a, n, invert);

    if (invert) {
        normalize<<<grid, block>>>(d_a, n);
    }

    hipMemcpy(h_a, d_a, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_temp);
}

void generateComplexData(hipDoubleComplex *a, int n) {
    for (int i = 0; i < n; ++i) {
        double real = static_cast<double>(rand()) / RAND_MAX;
        double imag = static_cast<double>(rand()) / RAND_MAX;
        a[i] = make_hipDoubleComplex(real, imag);
    }
}

int main() {
    int sizes[] = {256, 512, 1024, 2048, 4096, 8192, 16384, 32768, 65536, 131072, 262144, 524288, 1048576, 2097152, 4194304, 8388608};
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);

    hipFree(0);  // Initialize CUDA

    for (int i = 0; i < num_sizes; ++i) {
        int n = sizes[i];
        hipDoubleComplex *data = (hipDoubleComplex *)malloc(n * sizeof(hipDoubleComplex));

        generateComplexData(data, n);

        struct timeval start, end;
        gettimeofday(&start, NULL);
        
        fft(data, n, false);
        
        gettimeofday(&end, NULL);
        long seconds = (end.tv_sec - start.tv_sec);
        long micros = ((seconds * 1000000) + end.tv_usec) - (start.tv_usec);

        printf("FFT size %d - Execution time: %ld microseconds\n", n, micros);

        free(data);
    }

    return 0;
}
